#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include "wb.h"
using namespace std;

#define CEIL(a, b) ((a-1)/b +1)

__global__ void RGB_to_Gray(float *inputImage, float *outputImage, int height, int width) {

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x>=height || y>=width)
		return;

	unsigned int idx = x * width + y;
	float r = inputImage[3 * idx];	
	float g = inputImage[3 * idx + 1];
	float b = inputImage[3 * idx + 2];
	outputImage[idx] = (0.21f * r + 0.71f * g + 0.07f * b);
}

int main(int argc, char *argv[]) {

	int imageChannels;
	int imageWidth;
	int imageHeight;
	char *inputImageFile;
	wbImage_t inputImage;
	wbImage_t outputImage;
	float *hostInputImageData;
	float *hostOutputImageData;
	float *deviceInputImageData;
	float *deviceOutputImageData;

	/* parse the input arguments */
	wbArg_t args = wbArg_read(argc, argv);

	inputImageFile = wbArg_getInputFile(args, 0);
	inputImage = wbImport(inputImageFile);

	imageWidth  = wbImage_getWidth(inputImage);
	imageHeight = wbImage_getHeight(inputImage);
	imageChannels = wbImage_getChannels(inputImage); // For this lab the value is always 3

	// Since the image is monochromatic, it only contains one channel
	outputImage = wbImage_new(imageWidth, imageHeight, 1);

	hostInputImageData  = wbImage_getData(inputImage);
	hostOutputImageData = wbImage_getData(outputImage);

	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

	wbTime_start(GPU, "Doing GPU memory allocation");
	hipMalloc((void **)&deviceInputImageData,
		imageWidth * imageHeight * imageChannels * sizeof(float));
	hipMalloc((void **)&deviceOutputImageData,
		imageWidth * imageHeight * sizeof(float));
	wbTime_stop(GPU, "Doing GPU memory allocation");

	wbTime_start(Copy, "Copying data to the GPU");
	hipMemcpy(deviceInputImageData, hostInputImageData,
		imageWidth * imageHeight * imageChannels * sizeof(float), hipMemcpyHostToDevice);
	wbTime_stop(Copy, "Copying data to the GPU");

	///////////////////////////////////////////////////////
	wbTime_start(Compute, "Doing the computation on the GPU");
	dim3 block(32, 32, 1);
	dim3 grid(CEIL(imageHeight, 32), CEIL(imageWidth, 32), 1);

	RGB_to_Gray <<<grid, block>>> (deviceInputImageData, deviceOutputImageData, 
		imageHeight, imageWidth);

	wbTime_stop(Compute, "Doing the computation on the GPU");

	///////////////////////////////////////////////////////
	wbTime_start(Copy, "Copying data from the GPU");
	hipMemcpy(hostOutputImageData, deviceOutputImageData,
		imageWidth * imageHeight * sizeof(float), hipMemcpyDeviceToHost);
	wbTime_stop(Copy, "Copying data from the GPU");

	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

	wbSolution(args, outputImage);

	hipFree(deviceInputImageData);
	hipFree(deviceOutputImageData);

	wbImage_delete(outputImage);
	wbImage_delete(inputImage);

	return 0;
}
