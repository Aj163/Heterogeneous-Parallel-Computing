#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

#define N ((int)1e4)
#define CEIL(a, b) ((a-1)/b +1)

__global__ void add(int *d_a, int *d_b, int *d_c) {

	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if(x < N && y < N)
		d_c[x*N + y] = d_b[x*N + y] + d_a[x*N + y];
}

int main() {

	int *h_a, *h_b, *h_c;
	int *d_a, *d_b, *d_c;

	// Allocate host memory
	h_a = new int[N*N];
	h_b = new int[N*N];
	h_c = new int[N*N];

	printf("\nSize of matrices : %d x %d\n\n", N, N);

	srand(time(0));
	for(int i=0; i<N; i++)
		for(int j=0; j<N; j++) {
			h_a[i*N + j] = rand()%N;
			h_b[i*N + j] = rand()%N;
		}

	// Allocate memory on device
	hipMalloc((void**)&d_a, N*N*sizeof(int));
	hipMalloc((void**)&d_b, N*N*sizeof(int));
	hipMalloc((void**)&d_c, N*N*sizeof(int));

	//Copy data into device memory
	hipMemcpy(d_a, h_a, N*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N*N*sizeof(int), hipMemcpyHostToDevice);

	// Kernel call
	dim3 grid(CEIL(N, 32), CEIL(N, 32), 1);
	dim3 block(32, 32, 1);

	add<<<grid, block>>>(d_a, d_b, d_c);
	hipDeviceSynchronize();

	// Copy data back to host
	hipMemcpy(h_c, d_c, N*N*sizeof(int), hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	int errors = 0;
	for(int i=0; i<N; i++)
		for(int j=0; j<N; j++)
			if(h_c[i*N + j] != h_a[i*N + j] + h_b[i*N + j]) {
				errors++;
				if(errors <= 10)
					printf("Test failed at (%d, %d)\n", i, j);
			}

	if(errors)
		printf("\n%d Tests failed!\n\n", errors);
	else
		printf("All tests passed !\n\n");	

	// Free host memory
	delete[] h_a, h_b, h_c;
}