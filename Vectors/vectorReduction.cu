#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

#define N ((int)1e7)
#define CEIL(a, b) ((a-1)/b +1)

__global__ void reduce(int *d_a, int *sum) {

	__shared__ int data[1024];
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	// Copy all elements in block to shared memory and wait
	data[threadIdx.x] = d_a[i];
	__syncthreads();

	for(int step=1; step<1024; step*=2) {
		int threadID = 2*step*threadIdx.x;
		if(threadID + step < 1024)
			data[threadID] += data[threadID + step];

		__syncthreads();
	}

	if(threadIdx.x == 0)
		atomicAdd(sum, data[0]);
}

int main() {

	int *h_a, *h_sum;
	int *d_a, *d_sum;
	clock_t tim;

	h_a = new int[N];
	h_sum = new int;
	*h_sum = 0;

	printf("\nValue of N   : %d\n", N);

	srand(time(0));
	for(int i=0; i<N; i++)
		h_a[i] = rand()%2;

	hipMalloc((void**)&d_a, N*sizeof(int));
	hipMalloc((void**)&d_sum, sizeof(int));

	hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_sum, h_sum, sizeof(int), hipMemcpyHostToDevice);

	// Device timer	
	hipEvent_t start, stop;
	float tims;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Kernel call
	reduce<<<CEIL(N, 1024), 1024>>>(d_a, d_sum);
	hipDeviceSynchronize();

	// End timer
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&tims, start, stop);
	printf("\nDevice Time : %0.2lf ms\n", tims);

	hipMemcpy(h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_sum);

	int sum = 0;
	tim = clock();
	for(int i=0; i<N; i++)
		sum += h_a[i];
	tim = clock() - tim;
	printf("Host Time   : %0.2lf ms\n", tim*1.0/CLOCKS_PER_SEC*1000.0);

	printf("\nDevice sum  : %d\nHost sum    : %d\n\n", *h_sum, sum);

	delete[] h_a;
	delete h_sum;
}	