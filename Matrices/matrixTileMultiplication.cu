#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

#define N ((int)1e3)
#define TILE 32
#define CEIL(a, b) ((a-1)/b +1)

__global__ void multiply(float *d_a, float *d_b, float *d_c) {

	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	__shared__ float a[TILE][TILE];
	__shared__ float b[TILE][TILE];

	float cij = 0.0;
	for(int k=0; k<CEIL(N, TILE); k++) {
		// Copy the kth tile on the horizontal strip from A
		if(x<N && TILE*k + threadIdx.y < N)
			a[threadIdx.x][threadIdx.y] = d_a[x*N + TILE*k + threadIdx.y];
		else
			a[threadIdx.x][threadIdx.y] = 0.0;

		// Copy the kth tile on the vertical strip from B
		if(y<N && TILE*k + threadIdx.x < N)
			b[threadIdx.x][threadIdx.y] = d_b[(TILE*k + threadIdx.x)*N + y];
		else
			b[threadIdx.x][threadIdx.y] = 0.0;

		// Wait for all elements to be copied
		__syncthreads();

		// Do all operations related to these tiles before moving to next tile
		for(int kk=0; kk<TILE; kk++)
			cij += a[threadIdx.x][kk] * b[kk][threadIdx.y];

		// Wait before copying next tile
		__syncthreads();
	}

	if(x < N && y < N)
		d_c[x*N + y] = cij;
}

template <class T>
void testSolution(T *h_a, T *h_b, T *h_c, float precision=0.0) {

	int errors = 0;
	for(int i=0; i<N; i++)
		for(int j=0; j<N; j++) {

			float exp = 0.0, act;
			for(int k=0; k<N; k++)
				exp += h_a[i*N + k] * h_b[k*N + j];
			act = h_c[i*N + j];

			if(abs(act-exp) / (max(exp, precision)) > precision) {
				
				errors++;
				if(errors <= 10)
					printf("Test failed at index : (%d, %d) [Expected: %10.2f | Got: %10.2f]\n", 
						i, j, exp, act);
			}
		}

	if(errors)
		printf("\n%d Tests failed!\n\n", errors);
	else
		printf("All tests passed !\n\n");
}

int main() {

	float *h_a, *h_b, *h_c;
	float *d_a, *d_b, *d_c;

	// Allocate host memory
	h_a = new float[N*N];
	h_b = new float[N*N];
	h_c = new float[N*N];

	printf("\nSize of matrices : %d x %d\n\n", N, N);

	srand(time(0));
	for(int i=0; i<N; i++)
		for(int j=0; j<N; j++) {
			h_a[i*N + j] = (rand()%N) *1.0/ (rand()%N +1);
			h_b[i*N + j] = (rand()%N) *1.0/ (rand()%N +1);
		}

	// Allocate memory on device
	hipMalloc((void**)&d_a, N*N*sizeof(float));
	hipMalloc((void**)&d_b, N*N*sizeof(float));
	hipMalloc((void**)&d_c, N*N*sizeof(float));

	//Copy data into device memory
	hipMemcpy(d_a, h_a, N*N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N*N*sizeof(float), hipMemcpyHostToDevice);

	// Kernel call
	dim3 grid(CEIL(N, 32), CEIL(N, 32), 1);
	dim3 block(32, 32, 1);

	multiply <<<grid, block>>> (d_a, d_b, d_c);
	hipDeviceSynchronize();

	// Copy data back to host
	hipMemcpy(h_c, d_c, N*N*sizeof(float), hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	testSolution(h_a, h_b, h_c, 1e-3); //Tolerates 0.1% relative error

	// Free host memory
	delete[] h_a, h_b, h_c;
}