#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

#define N (1<<30)

__global__ void add(float *d_a, float *d_b, float *d_c) {

	int i = blockIdx.x;
	if(i < N)
		d_c[i] = d_b[i] + d_a[i];
}

int main() {

	float *h_a, *h_b, *h_c;
	float *d_a, *d_b, *d_c;
	clock_t tim;

	// Allocate host memory
	h_a = new float[N];
	h_b = new float[N];
	h_c = new float[N];

	printf("Number of elements in array : %d\n", N);

	srand(time(0));
	for(int i=0; i<2; i++) {
		h_a[i] = rand()%2;
		h_b[i] = rand()%2;
	}

	// Allocate memory on device
	hipMalloc((void**)&d_a, N*sizeof(float));
	hipMalloc((void**)&d_b, N*sizeof(float));
	hipMalloc((void**)&d_c, N*sizeof(float));

	//Copy data into device memory
	hipMemcpy(d_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N*sizeof(float), hipMemcpyHostToDevice);

	
	// Create a timer for device
	hipEvent_t start, stop;
	float tims;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Kernel call
	add<<<N, 1>>>(d_a, d_b, d_c);
	hipDeviceSynchronize();

	// Stop times=r
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&tims, start, stop);
	printf("Device Time: %0.2lf us\n", 1000*tims);

	// Copy data back to host
	hipMemcpy(h_c, d_c, N*sizeof(float), hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	tim = clock();
	for(int i=0; i<N; i++)
		h_c[i] = h_a[i] + h_b[i];
	tim = clock() - tim;
	printf("Host Time: %0.2lf s\n", tim*1.0/CLOCKS_PER_SEC);

	// Free host memory
	delete[] h_a, h_b, h_c;
}